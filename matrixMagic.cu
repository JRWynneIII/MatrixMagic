#include "hip/hip_runtime.h"
#include <iostream>
#include "matrixMagic.h"
using namespace std;

__global__ void transposeKernel(double* internal_storage)
{
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double temp = internal_storage[tid];
  internal_storage[tid] = internal_storage[threadIdx.x * blockDim.x + blockIdx.x];
  internal_storage[threadIdx.x * blockDim.x + blockIdx.x] = temp;
}

void transposeWrapper(double* internal_storage, int m, int n)
{
  double* d_storage;
  hipMalloc(&d_storage, m*n*sizeof(double));
  hipMemcpy(d_storage, internal_storage, m*n*sizeof(double), hipMemcpyHostToDevice);
  transposeKernel<<<n,m>>>(d_storage);
  hipMemcpy(internal_storage, d_storage, m*n*sizeof(double), hipMemcpyDeviceToHost);
}
