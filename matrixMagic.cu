#include "hip/hip_runtime.h"
#include <iostream>
#include "matrixMagic.h"
using namespace std;

#define internal_storage(r, c) (internal_storage[(r)*n + (c)])
#define output(r, c) (output[(r)*n + (c)])

void CUDAErrorCheck()
{
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    cout << "CUDA error : " << hipGetErrorString(error) << " (" << error << ")\n";
    exit(EXIT_FAILURE);
  }
}

__global__ void transposeKernel(double* internal_storage, int m, int n, double* output)
{
  int tid = threadIdx.x;
  for(int j = tid+1; j < n; j++)
  {
    double temp = internal_storage(tid,j);
    internal_storage(tid, j) = internal_storage(j,tid);
    internal_storage(j,tid) = temp;
  }
}

void transposeWrapper(double* internal_storage, int m, int n)
{
  double* d_storage;
  double* d_output;

  hipMalloc(&d_storage, m*n*sizeof(double));
  hipMalloc(&d_output, m*n*sizeof(double));
  hipMemcpy(d_storage, internal_storage, m*n*sizeof(double), hipMemcpyHostToDevice);
  CUDAErrorCheck();
  transposeKernel<<<1,m*n>>>(d_storage,m,n,d_output);
  hipMemcpy(internal_storage, d_storage, m*n*sizeof(double), hipMemcpyDeviceToHost);
  CUDAErrorCheck();
}
